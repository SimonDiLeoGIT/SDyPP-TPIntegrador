
#include <hip/hip_runtime.h>
#include <stdint.h>
#include <stdio.h>

__device__ uint32_t shifts[] = {  7, 12, 17, 22,  5,  9, 14, 20,  4, 11, 16, 23,  6, 10, 15, 21 };
__device__ uint32_t sines[]  = { 0xd76aa478, 0xe8c7b756, 0x242070db, 0xc1bdceee, 0xf57c0faf, 0x4787c62a, 0xa8304613, 0xfd469501,
                                 0x698098d8, 0x8b44f7af, 0xffff5bb1, 0x895cd7be, 0x6b901122, 0xfd987193, 0xa679438e, 0x49b40821,
                                 0xf61e2562, 0xc040b340, 0x265e5a51, 0xe9b6c7aa, 0xd62f105d, 0x02441453, 0xd8a1e681, 0xe7d3fbc8,
                                 0x21e1cde6, 0xc33707d6, 0xf4d50d87, 0x455a14ed, 0xa9e3e905, 0xfcefa3f8, 0x676f02d9, 0x8d2a4c8a,
                                 0xfffa3942, 0x8771f681, 0x6d9d6122, 0xfde5380c, 0xa4beea44, 0x4bdecfa9, 0xf6bb4b60, 0xbebfbc70,
                                 0x289b7ec6, 0xeaa127fa, 0xd4ef3085, 0x04881d05, 0xd9d4d039, 0xe6db99e5, 0x1fa27cf8, 0xc4ac5665,
                                 0xf4292244, 0x432aff97, 0xab9423a7, 0xfc93a039, 0x655b59c3, 0x8f0ccc92, 0xffeff47d, 0x85845dd1,
                                 0x6fa87e4f, 0xfe2ce6e0, 0xa3014314, 0x4e0811a1, 0xf7537e82, 0xbd3af235, 0x2ad7d2bb, 0xeb86d391 };

__device__ uint32_t left_rotate(uint32_t x, uint32_t c) {
    return ((x << c) | (x >> (32 - c)));
}

__device__ void cuda_md5(const uint8_t* initial_msg, size_t initial_len, uint8_t* digest) {
    uint32_t h0, h1, h2, h3;

    // These vars will contain the hash
    h0 = 0x67452301;
    h1 = 0xefcdab89;
    h2 = 0x98badcfe;
    h3 = 0x10325476;

    int new_len = ((((initial_len + 8) / 64) + 1) * 64) - 8;

    uint8_t* msg = (uint8_t*)malloc(new_len + 64);
    memcpy(msg, initial_msg, initial_len);
    msg[initial_len] = 128; // append the "1" bit; most significant bit is "1"
    for (int i = initial_len + 1; i < new_len; i++) {
        msg[i] = 0; // append "0" bits
    }

    uint64_t bits_len = 8 * initial_len;
    memcpy(msg + new_len, &bits_len, 8); // in little-endian

    for (int offset = 0; offset < new_len; offset += 64) {
        uint32_t* w = (uint32_t*)(msg + offset);
        uint32_t a = h0, b = h1, c = h2, d = h3;

        for (int i = 0; i < 64; i++) {
            uint32_t f, g;

            if (i < 16) {
                f = (b & c) | ((~b) & d);
                g = i;
            } else if (i < 32) {
                f = (d & b) | ((~d) & c);
                g = (5 * i + 1) % 16;
            } else if (i < 48) {
                f = b ^ c ^ d;
                g = (3 * i + 5) % 16;
            } else {
                f = c ^ (b | (~d));
                g = (7 * i) % 16;
            }

            uint32_t temp = d;
            d = c;
            c = b;
            b = b + left_rotate((a + f + sines[i] + w[g]), shifts[(i / 16) * 4 + (i % 4)]);
            a = temp;
        }

        h0 += a;
        h1 += b;
        h2 += c;
        h3 += d;
    }

    free(msg);

    ((uint32_t*)digest)[0] = h0;
    ((uint32_t*)digest)[1] = h1;
    ((uint32_t*)digest)[2] = h2;
    ((uint32_t*)digest)[3] = h3;
}