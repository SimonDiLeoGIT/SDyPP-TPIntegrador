
#include <hip/hip_runtime.h>
#include <stdio.h>

#define gpuErrchk(ans) { gpuAssert(ans); }
inline void gpuAssert(hipError_t code) {
	if (code != hipSuccess) {
    	fprintf(stderr, "GPUassert:%s\n",
                     hipGetErrorString(code));
    	exit(code);
	}
}

int main() {
    gpuErrchk( hipDeviceSynchronize() );
	return 0;
}